#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <chrono>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

void print(int *mat, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        cout << mat[i] << " ";
    }

    return;
}

void initData(int *mat, const int size)
{
    int i;

    srand (time(0));
    for(i = 0; i < size; i++)
    {
        mat[i] = rand() % 10 + 1;
    }

    return;
}

void multMatrixOnHost(int *A, int *B, int *C, const int cols,
                     const int rows)
{
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            C[j * rows + i] = 0;
            for (int shared_dim = 0; shared_dim < cols; shared_dim++)
            {
                //dot product
                C[j * rows + i] += A[shared_dim * rows + i] * B[j * rows + shared_dim];
            }
        }
    }

    return;
}

void checkResult(int *hostRef, int *gpuRef, const int N)
{
    double epsilon = 1.0E-8;
    bool match = 1;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            match = 0;
            printf("host %d gpu %d\n", hostRef[i], gpuRef[i]);
            break;
        }
    }

    if (match)
        printf("Arrays match.\n\n");
    else
        printf("Arrays do not match.\n\n");
}

// grid 1D block 1D
__global__ void multMatrixOnGPU1D(int *A, int *B, int *C, const int cols,
                     const int rows)
{
    unsigned int ix_cols = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix_cols < cols )
        for (int iy_rows = 0; iy_rows < rows; iy_rows++)
        {
            int idx = iy_rows * cols + ix_cols;
            C[idx] = 0;
            for (int shared_dim = 0; shared_dim < cols; shared_dim++)
            {
                //dot product
                C[idx] += A[shared_dim * rows + ix_cols] * B[iy_rows * rows + shared_dim];
            }
        }
}

int main(int argc, char **argv)
{
    // set up data size of matrix
    int nx = 0;
    int ny = 0;

    if(argc < 2)
    {
        nx = ny = 2;
    }
    else
    {
        nx = ny = stoi(argv[1]);
    }

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(int);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // malloc host memory
    int *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (int *)malloc(nBytes);
    h_B = (int *)malloc(nBytes);
    hostRef = (int *)malloc(nBytes);
    gpuRef = (int *)malloc(nBytes);

    // initialize data at host side

    initData(h_A, nxy);
    initData(h_B, nxy);
    multMatrixOnHost(h_A, h_B, hostRef, nx, ny);


    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result SAFE_CALLs
    auto start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnHost(h_A, h_B, hostRef, nx, ny);
    auto end_cpu =  chrono::high_resolution_clock::now();
    chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;

    printf("multMatrixOnHost elapsed %f ms\n", duration_ms.count());

    // malloc device global memory
    int *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // transfer data from host to device
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // invoke kernel at host side
    int dimx = 256;
    dim3 block(dimx, 1);
    dim3 grid((nx + block.x - 1) / block.x, 1);

    start_cpu =  chrono::high_resolution_clock::now();
    multMatrixOnGPU1D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
    end_cpu =  chrono::high_resolution_clock::now();

    duration_ms = end_cpu - start_cpu;

    printf("multMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %f ms\n", grid.x,
           grid.y,
           block.x, block.y, duration_ms.count());

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");


    return (0);
}
